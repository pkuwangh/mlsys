#include "hip/hip_runtime.h"
#include <functional>
#include <string>
#include <vector>

#include "matmul_kernel_basic.cuh"
#include "matmul_kernel_shmem_basic.cuh"
#include "matmul_utils.cuh"

// dump cuda-related device information
void dumpDeviceInfo() {
    int device_count;
    checkCuda(hipGetDeviceCount(&device_count), "hipGetDeviceCount");
    std::printf("Device count: %d\n", device_count);
    hipDeviceProp_t prop;
    checkCuda(hipGetDeviceProperties(&prop, 0), "hipGetDeviceProperties");
    std::printf("Device name: %s\n", prop.name);
    std::printf("Device compute capability: %d.%d\n", prop.major, prop.minor);
    std::printf("Device total memory: %zu bytes\n", prop.totalGlobalMem);
    // dump factors that limit the number of blocks
    std::printf("max threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
    std::printf("max blocks per SM: %d\n", prop.maxBlocksPerMultiProcessor);
    std::printf("# of registers per SM: %d\n", prop.regsPerMultiprocessor);
    std::printf("shared memory per SM: %zu bytes\n", prop.sharedMemPerMultiprocessor);
}

// sanity tests on basic kernel with small matrix sizes
void sanityTests() {
    std::printf("Sanity functional correctness check\n");
    MatmulBuffers buffers = MatmulBuffers(2, 4, 2);
    printMatrix(buffers.hA.data(), buffers.M, buffers.K, "A");
    printMatrix(buffers.hB.data(), buffers.K, buffers.N, "B");

    std::printf("basic kernel\n");
    buffers.reset();
    runMatmulBasic(buffers);
    buffers.printResult();

    // std::printf("shmem-basic kernel\n");
    // buffers.reset();
    // runMatmulShmemBasic(buffers);
    // buffers.printResult();

    std::printf("--------------------------------\n");
}

// verify correctness against reference result
void verifyCorrectness(const std::vector<float> &ref, MatmulBuffers &buffers, std::string kernel_name) {
    std::vector<float> result = buffers.copyResultVector();
    int num_errors = 0;
    for (int i = 0; i < result.size(); i++) {
        float error = std::abs(result[i] - ref[i]);
        if (error > 1.0) {
            std::printf("Error at index %d: %f != %f\n", i, result[i], ref[i]);
            ++ num_errors;
            if (num_errors >= 4) {
                break;
            }
        }
    }
    std::printf("%s correctness check %s\n", kernel_name.c_str(), num_errors == 0 ? "passed" : "failed");
}

// functional tests against basic kernel
void functionalTests() {
    std::printf("Functional correctness check against basic kernel\n");
    MatmulBuffers buffers = MatmulBuffers(32, 64, 64);
    runMatmulBasic(buffers);
    std::vector<float> ref = buffers.copyResultVector();
    verifyCorrectness(ref, buffers, "basic");

    // verify shmem-basic kernel
    buffers.reset();
    runMatmulShmemBasic(buffers);
    verifyCorrectness(ref, buffers, "shmem-basic");

    std::printf("--------------------------------\n");
}

// performance tests
void perfTests(MatmulBuffers &buffers, std::function<void(MatmulBuffers &)> run_kernel, std::string kernel_name) {
    int num_warmup_iters = 1;
    int num_total_iters = 100;
    DeviceTimer timer;
    for (int i = 0; i < num_warmup_iters; i++) {
        run_kernel(buffers);
    }
    buffers.reset();
    timer.start();
    for (int i = 0; i < num_total_iters; i++) {
        run_kernel(buffers);
    }
    checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize");
    buffers.printTFLOPS(timer.stop(), kernel_name);
}

int main() {
    dumpDeviceInfo();

    // sanity functional correctness check on basic kernel
    sanityTests();

    // verify correctness against basic kernel
    functionalTests();

    // perf test
    MatmulBuffers buffers = MatmulBuffers(4096, 8192, 8192);
    perfTests(buffers, runMatmulBasic, "matmul-basic");
    perfTests(buffers, runMatmulShmemBasic, "matmul-shmem-basic");

    return 0;
}
