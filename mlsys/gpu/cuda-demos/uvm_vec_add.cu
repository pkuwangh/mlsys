#include "hip/hip_runtime.h"
#include <boost/program_options.hpp>
#include <cmath>
#include <cstdint>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <set>
#include <string>

// CUDA error handling
#define CU_ASSERT_RESULT(x)                                                    \
    do {                                                                       \
        hipError_t cuResult = (x);                                               \
        if ((cuResult) != hipSuccess) {                                      \
            const char *errDescStr, *errNameStr;                               \
            hipDrvGetErrorString(cuResult, &errDescStr);                           \
            hipDrvGetErrorName(cuResult, &errNameStr);                             \
            fprintf(stderr, "[%s] %s in expression %s in %s() : %s:%d\n",      \
                    errNameStr, errDescStr, #x, __PRETTY_FUNCTION__, __FILE__, \
                    __LINE__);                                                 \
            std::exit(1);                                                      \
        }                                                                      \
    } while (0)

namespace po = boost::program_options;

// Kernel function to add the elements of two arrays
__global__ void add(size_t n, float *x, float *y) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}

int benchmark(const std::string &allocator) {
    size_t N = (size_t)(2) << 30; // 2G x 4B x 2
    size_t allocSize = N * sizeof(float);
    float *x, *y;
    hipMemGenericAllocationHandle_t handleX, handleY;

    if (allocator == "hipMallocManaged") {
        hipMallocManaged(&x, N * sizeof(float));
        hipMallocManaged(&y, N * sizeof(float));
    } else if (allocator == "hipHostMalloc") {
        hipHostMalloc(&x, N * sizeof(float));
        hipHostMalloc(&y, N * sizeof(float));
    } else if (allocator == "hipMemCreate-Device" ||
               allocator == "hipMemCreate-Host") {
        // get device handle
        int cudaDev;
        hipDevice_t currentDev;
        hipGetDevice(&cudaDev);
        hipDeviceGet(&currentDev, cudaDev);
        std::cout << "get cuda device " << cudaDev << "/" << currentDev
                  << std::endl;
        // get cpu NUMA id and set location type
        int cpuNumaNodeId = -1;
        hipMemLocationType type = hipMemLocationTypeDevice;
        hipDeviceGetAttribute(&cpuNumaNodeId, CU_DEVICE_ATTRIBUTE_HOST_NUMA_ID,
                             currentDev);
        bool hostMem =
            (cpuNumaNodeId != -1) && (allocator == "hipMemCreate-Host");
        type = hostMem ? hipMemLocationTypeHostNuma : type;
        std::cout << "hostMem-" << hostMem << ", host numa ID=" << cpuNumaNodeId
                  << std::endl;
        // memory allocation property
        hipMemAllocationProp memprop = {};
        memprop.type = hipMemAllocationTypePinned;
        memprop.location.type = type;
        memprop.location.id = hostMem ? cpuNumaNodeId : currentDev;
        // size & granularity
        size_t allocSize = N * sizeof(float);
        size_t granu = 0;
        hipMemGetAllocationGranularity(&granu, &memprop,
                                      hipMemAllocationGranularityMinimum);
        size_t origSize = allocSize;
        if (allocSize % granu > 0) {
            allocSize = granu * (allocSize / granu + 1);
        }
        std::cout << "requested size=" << origSize
                  << ", padded alloc size=" << allocSize
                  << ", granularity=" << granu << std::endl;
        // physical memory allocation
        CU_ASSERT_RESULT(hipMemCreate(&handleX, allocSize, &memprop, 0));
        CU_ASSERT_RESULT(hipMemCreate(&handleY, allocSize, &memprop, 0));
        // reserve an address space and map it to a pointer
        CU_ASSERT_RESULT(
            hipMemAddressReserve((hipDeviceptr_t *)(&x), allocSize, 0, 0, 0));
        CU_ASSERT_RESULT(
            hipMemAddressReserve((hipDeviceptr_t *)(&y), allocSize, 0, 0, 0));
        CU_ASSERT_RESULT(hipMemMap((hipDeviceptr_t)x, allocSize, 0, handleX, 0));
        CU_ASSERT_RESULT(hipMemMap((hipDeviceptr_t)y, allocSize, 0, handleY, 0));
        // explicitly protect mapped VA ranges
        hipMemAccessDesc accessDesc[2] = {{}};
        accessDesc[0].location.type = hipMemLocationTypeDevice;
        accessDesc[0].location.id = currentDev;
        accessDesc[0].flags = hipMemAccessFlagsProtReadWrite;
        accessDesc[1].location.type = type;
        accessDesc[1].location.id = hostMem ? cpuNumaNodeId : currentDev;
        accessDesc[1].flags = hipMemAccessFlagsProtReadWrite;
        CU_ASSERT_RESULT(hipMemSetAccess((hipDeviceptr_t)x, allocSize, accessDesc,
                                        hostMem ? 2 : 1));
        CU_ASSERT_RESULT(hipMemSetAccess((hipDeviceptr_t)y, allocSize, accessDesc,
                                        hostMem ? 2 : 1));
    } else if (allocator == "malloc") {
        x = (float *)(malloc(N * sizeof(float)));
        y = (float *)(malloc(N * sizeof(float)));
    } else {
        std::cout << "Unknown allocator " << allocator << std::endl;
        return -1;
    }

    float *x_cpu, *y_cpu;
    if (allocator == "hipMemCreate-Device") {
        x_cpu = (float *)(malloc(N * sizeof(float)));
        y_cpu = (float *)(malloc(N * sizeof(float)));
        for (size_t i = 0; i < N; ++i) {
            x_cpu[i] = 1.0f;
            y_cpu[i] = 2.0f;
        }
        hipMemcpy(x, x_cpu, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(y, y_cpu, N * sizeof(float), hipMemcpyHostToDevice);
    } else {
        for (size_t i = 0; i < N; ++i) {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }
    }

    hipEvent_t ckpt1, ckpt2, ckpt3;
    hipEventCreate(&ckpt1);
    hipEventCreate(&ckpt2);
    hipEventCreate(&ckpt3);

    std::cout << "Initialized " << N << " elements using " << allocator
              << std::endl;

    size_t blockSize = 1024;
    size_t numBlocks = (N + blockSize - 1) / blockSize;

    hipEventRecord(ckpt1);

    add<<<numBlocks, blockSize>>>(N, x, y);
    hipEventRecord(ckpt2);

    for (int k = 0; k < 10; ++k) {
        add<<<numBlocks, blockSize>>>(N, x, y);
    }
    hipEventRecord(ckpt3);

    hipEventSynchronize(ckpt3);
    // hipDeviceSynchronize();

    float elapsed = 0;
    hipEventElapsedTime(&elapsed, ckpt1, ckpt2);
    std::cout << "Elapsed time " << int(elapsed) << " mili-seconds - initial"
              << std::endl;
    hipEventElapsedTime(&elapsed, ckpt2, ckpt3);
    std::cout << "Elapsed time " << int(elapsed / 10)
              << " mili-seconds - warmedup" << std::endl;

    float maxError = 0.0f;
    if (allocator == "hipMemCreate-Device") {
        hipMemcpy(y_cpu, y, N * sizeof(float), hipMemcpyDeviceToHost);
        for (size_t i = 0; i < N; i++) {
            maxError = fmax(maxError, fabs(y_cpu[i] - 13.0f));
        }
    } else {
        for (size_t i = 0; i < N; i++) {
            maxError = fmax(maxError, fabs(y[i] - 13.0f));
        }
    }

    hipEventDestroy(ckpt1);
    hipEventDestroy(ckpt2);
    hipEventDestroy(ckpt3);

    if (allocator == "hipMemCreate") {
        hipMemAddressFree((hipDeviceptr_t)x, allocSize);
        hipMemAddressFree((hipDeviceptr_t)y, allocSize);
        hipMemRelease(handleX);
        hipMemRelease(handleY);
    } else if (allocator == "malloc") {
        free(x);
        free(y);
    } else {
        hipFree(x);
        hipFree(y);
    }
    if (allocator == "hipMemCreate-Device") {
        free(x_cpu);
        free(y_cpu);
    }
    return int(maxError);
}

int main(int argc, char **argv) {
    std::set<std::string> avail_allocators(
        {"hipHostMalloc", "hipMallocManaged", "hipMemCreate-Device",
         "hipMemCreate-Host", "malloc"});
    std::string allocator;
    po::options_description all_opts("uvm_vec_add CLI");
    all_opts.add_options()("help,h", "Help message");
    all_opts.add_options()("list,l", "list available allocator options");
    all_opts.add_options()(
        "allocator,a", po::value<std::string>(&allocator)->default_value("all"),
        "Which memory allocation API to use");

    po::variables_map vm;
    try {
        po::store(po::parse_command_line(argc, argv, all_opts), vm);
        po::notify(vm);
    } catch (...) {
        std::cout << "Error: Invalid Arguments " << std::endl;
        std::cout << all_opts << std::endl;
        return 1;
    }
    if (vm.count("help")) {
        std::cout << all_opts << std::endl;
        return 0;
    }
    if (vm.count("list")) {
        for (const std::string &x : avail_allocators) {
            std::cout << x << std::endl;
        }
        return 0;
    }

    int maxError = 0;
    if (allocator == "all") {
        for (const std::string &x : avail_allocators) {
            std::cout << "\nBenchmarking " << x << std::endl;
            maxError = benchmark(x);
            std::cout << "Max error is " << maxError << std::endl;
        }
    } else {
        maxError = benchmark(allocator);
        std::cout << "Max error is " << maxError << std::endl;
    }

    return 0;
}
