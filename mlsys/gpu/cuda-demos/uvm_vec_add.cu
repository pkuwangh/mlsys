#include "hip/hip_runtime.h"
#include <boost/program_options.hpp>
#include <cmath>
#include <cstdint>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <set>
#include <string>

namespace po = boost::program_options;

// Kernel function to add the elements of two arrays
__global__ void add(size_t n, float *x, float *y) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}

int benchmark(const std::string &allocator) {
    size_t N = (size_t)(2) << 30; // 2G x 4B x 2
    size_t allocSize = N * sizeof(float);
    float *x, *y;
    hipMemGenericAllocationHandle_t handleX, handleY;

    if (allocator == "hipMallocManaged") {
        hipMallocManaged(&x, N * sizeof(float));
        hipMallocManaged(&y, N * sizeof(float));
    } else if (allocator == "hipHostMalloc") {
        hipHostMalloc(&x, N * sizeof(float));
        hipHostMalloc(&y, N * sizeof(float));
    } else if (allocator == "hipMemCreate") {
        // get device handle
        int cudaDev;
        hipDevice_t currentDev;
        hipGetDevice(&cudaDev);
        hipDeviceGet(&currentDev, cudaDev);
        std::cout << "get cuda device " << cudaDev << "/" << currentDev
                  << std::endl;
        // get cpu NUMA id and set location type
        int cpuNumaNodeId = -1;
        hipMemLocationType type = hipMemLocationTypeDevice;
        hipDeviceGetAttribute(&cpuNumaNodeId, CU_DEVICE_ATTRIBUTE_HOST_NUMA_ID,
                             currentDev);
        bool hostMem = (cpuNumaNodeId != -1);
        type = hostMem ? hipMemLocationTypeHostNuma : type;
        std::cout << "hostMem-" << hostMem << ", host numa ID=" << cpuNumaNodeId
                  << std::endl;
        // memory allocation property
        hipMemAllocationProp memprop = {};
        memprop.type = hipMemAllocationTypePinned;
        memprop.location.type = type;
        memprop.location.id = hostMem ? cpuNumaNodeId : currentDev;
        // size & granularity
        size_t allocSize = N * sizeof(float);
        size_t granu = 0;
        hipMemGetAllocationGranularity(&granu, &memprop,
                                      hipMemAllocationGranularityMinimum);
        size_t origSize = allocSize;
        if (allocSize % granu > 0) {
            allocSize = granu * (allocSize / granu + 1);
        }
        std::cout << "requested size=" << origSize
                  << ", padded alloc size=" << allocSize
                  << ", granularity=" << granu << std::endl;
        // physical memory allocation
        hipMemCreate(&handleX, allocSize, &memprop, 0);
        hipMemCreate(&handleY, allocSize, &memprop, 0);
        // reserve an address space and map it to a pointer
        hipMemAddressReserve((hipDeviceptr_t *)(&x), allocSize, 0, 0, 0);
        hipMemAddressReserve((hipDeviceptr_t *)(&y), allocSize, 0, 0, 0);
        hipMemMap((hipDeviceptr_t)x, allocSize, 0, handleX, 0);
        hipMemMap((hipDeviceptr_t)y, allocSize, 0, handleY, 0);
        // explicitly protect mapped VA ranges
        hipMemAccessDesc accessDesc[2] = {{}};
        accessDesc[0].location.type = hipMemLocationTypeDevice;
        accessDesc[0].location.id = currentDev;
        accessDesc[0].flags = hipMemAccessFlagsProtReadWrite;
        accessDesc[1].location.type = type;
        accessDesc[1].location.id = hostMem ? cpuNumaNodeId : currentDev;
        accessDesc[1].flags = hipMemAccessFlagsProtReadWrite;
        hipMemSetAccess((hipDeviceptr_t)x, allocSize, accessDesc, hostMem ? 2 : 1);
        hipMemSetAccess((hipDeviceptr_t)y, allocSize, accessDesc, hostMem ? 2 : 1);
    } else if (allocator == "malloc") {
        x = (float *)(malloc(N * sizeof(float)));
        y = (float *)(malloc(N * sizeof(float)));
    } else {
        std::cout << "Unknown allocator " << allocator << std::endl;
        return -1;
    }

    for (size_t i = 0; i < N; ++i) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipEvent_t ckpt1, ckpt2, ckpt3;
    hipEventCreate(&ckpt1);
    hipEventCreate(&ckpt2);
    hipEventCreate(&ckpt3);

    std::cout << "Initialized " << N << " elements using " << allocator
              << std::endl;

    size_t blockSize = 1024;
    size_t numBlocks = (N + blockSize - 1) / blockSize;

    hipEventRecord(ckpt1);

    add<<<numBlocks, blockSize>>>(N, x, y);
    hipEventRecord(ckpt2);

    for (int k = 0; k < 10; ++k) {
        add<<<numBlocks, blockSize>>>(N, x, y);
    }
    hipEventRecord(ckpt3);

    hipEventSynchronize(ckpt3);
    // hipDeviceSynchronize();

    float elapsed = 0;
    hipEventElapsedTime(&elapsed, ckpt1, ckpt2);
    std::cout << "Elapsed time " << int(elapsed) << " mili-seconds - initial"
              << std::endl;
    hipEventElapsedTime(&elapsed, ckpt2, ckpt3);
    std::cout << "Elapsed time " << int(elapsed / 10)
              << " mili-seconds - warmedup" << std::endl;

    float maxError = 0.0f;
    for (size_t i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 103.0f));
    }

    hipEventDestroy(ckpt1);
    hipEventDestroy(ckpt2);
    hipEventDestroy(ckpt3);

    if (allocator == "hipMemCreate") {
        hipMemAddressFree((hipDeviceptr_t)x, allocSize);
        hipMemAddressFree((hipDeviceptr_t)y, allocSize);
        hipMemRelease(handleX);
        hipMemRelease(handleY);
    } else if (allocator == "malloc") {
        free(x);
        free(y);
    } else {
        hipFree(x);
        hipFree(y);
    }
    return int(maxError);
}

int main(int argc, char **argv) {
    std::set<std::string> avail_allocators(
        {"hipHostMalloc", "hipMallocManaged", "hipMemCreate", "malloc"});
    std::string allocator;
    po::options_description all_opts("uvm_vec_add CLI");
    all_opts.add_options()("help,h", "Help message");
    all_opts.add_options()("list,l", "list available allocator options");
    all_opts.add_options()(
        "allocator,a", po::value<std::string>(&allocator)->default_value("all"),
        "Which memory allocation API to use");

    po::variables_map vm;
    try {
        po::store(po::parse_command_line(argc, argv, all_opts), vm);
        po::notify(vm);
    } catch (...) {
        std::cout << "Error: Invalid Arguments " << std::endl;
        std::cout << all_opts << std::endl;
        return 1;
    }
    if (vm.count("help")) {
        std::cout << all_opts << std::endl;
        return 0;
    }
    if (vm.count("list")) {
        for (const std::string &x : avail_allocators) {
            std::cout << x << std::endl;
        }
        return 0;
    }

    int maxError = 0;
    if (allocator == "all") {
        for (const std::string &x : avail_allocators) {
            maxError = benchmark(x);
            std::cout << "Max error is " << maxError << std::endl;
        }
    } else {
        maxError = benchmark(allocator);
        std::cout << "Max error is " << maxError << std::endl;
    }

    return 0;
}