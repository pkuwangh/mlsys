#include "hip/hip_runtime.h"
#include <boost/program_options.hpp>
#include <cmath>
#include <cstdint>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <set>
#include <string>

// CUDA error handling
#define CU_ASSERT_RESULT(x)                                                                                            \
    do {                                                                                                               \
        hipError_t cuResult = (x);                                                                                       \
        if ((cuResult) != hipSuccess) {                                                                              \
            const char *errDescStr, *errNameStr;                                                                       \
            hipDrvGetErrorString(cuResult, &errDescStr);                                                                   \
            hipDrvGetErrorName(cuResult, &errNameStr);                                                                     \
            fprintf(stderr, "[%s] %s in expr %s in %s() : %s:%d\n", errNameStr, errDescStr, #x, __PRETTY_FUNCTION__,   \
                    __FILE__, __LINE__);                                                                               \
            std::exit(EXIT_FAILURE);                                                                                   \
        }                                                                                                              \
    } while (0)

#define CU_ASSERT_ERROR(x)                                                                                             \
    do {                                                                                                               \
        hipError_t cuError = (x);                                                                                     \
        if ((cuError) != hipSuccess) {                                                                                \
            fprintf(stderr, "[%s] %s in expr %s in %s() : %s:%D\n", hipGetErrorName(cuError),                         \
                    hipGetErrorString(cuError), #x, __PRETTY_FUNCTION__, __FILE__, __LINE__);                         \
            std::exit(EXIT_FAILURE);                                                                                   \
        }                                                                                                              \
    } while (0)

namespace po = boost::program_options;

// Kernel function to add the elements of two arrays
__global__ void add(size_t n, float *x, float *y) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}

int benchmark(const std::string &allocator) {
    // size_t N = (size_t)(2) << 20; // 2M x 4B
    size_t N = (size_t)(2) << 30; // 2G x 4B
    size_t allocSize = N * sizeof(float);
    std::cout << "AllocSize=" << allocSize << " / " << (allocSize / 1024 / 1024) << "MB" << std::endl;
    float *x, *y;
    hipMemGenericAllocationHandle_t handleX, handleY;

    if (allocator == "hipMallocManaged") {
        CU_ASSERT_ERROR(hipMallocManaged(&x, allocSize));
        CU_ASSERT_ERROR(hipMallocManaged(&y, allocSize));
    } else if (allocator == "hipHostMalloc") {
        CU_ASSERT_ERROR(hipHostMalloc(&x, allocSize));
        CU_ASSERT_ERROR(hipHostMalloc(&y, allocSize));
    } else if (allocator == "hipMemCreate-Device" || allocator == "hipMemCreate-Host") {
        // get device handle
        int cudaDev;
        hipDevice_t currentDev;
        CU_ASSERT_ERROR(hipGetDevice(&cudaDev));
        CU_ASSERT_RESULT(hipDeviceGet(&currentDev, cudaDev));
        std::cout << "get cuda device " << cudaDev << "/" << currentDev << std::endl;
        // get cpu NUMA id and set location type
        int cpuNumaNodeId = -1;
        hipMemLocationType type = hipMemLocationTypeDevice;
        CU_ASSERT_RESULT(hipDeviceGetAttribute(&cpuNumaNodeId, CU_DEVICE_ATTRIBUTE_HOST_NUMA_ID, currentDev));
        bool hostMem = (cpuNumaNodeId != -1) && (allocator == "hipMemCreate-Host");
        type = hostMem ? hipMemLocationTypeHostNuma : type;
        std::cout << "hostMem-" << hostMem << ", host numa ID=" << cpuNumaNodeId << std::endl;
        // memory allocation property
        hipMemAllocationProp memprop = {};
        memprop.type = hipMemAllocationTypePinned;
        memprop.location.type = type;
        memprop.location.id = hostMem ? cpuNumaNodeId : currentDev;
        // size & granularity
        size_t granu = 0;
        CU_ASSERT_RESULT(hipMemGetAllocationGranularity(&granu, &memprop, hipMemAllocationGranularityMinimum));
        size_t origSize = allocSize;
        if (allocSize % granu > 0) {
            allocSize = granu * (allocSize / granu + 1);
        }
        std::cout << "requested size=" << origSize << ", padded alloc size=" << allocSize << ", granularity=" << granu
                  << std::endl;
        // physical memory allocation
        CU_ASSERT_RESULT(hipMemCreate(&handleX, allocSize, &memprop, 0));
        CU_ASSERT_RESULT(hipMemCreate(&handleY, allocSize, &memprop, 0));
        // reserve an address space and map it to a pointer
        CU_ASSERT_RESULT(hipMemAddressReserve((hipDeviceptr_t *)(&x), allocSize, 0, 0, 0));
        CU_ASSERT_RESULT(hipMemAddressReserve((hipDeviceptr_t *)(&y), allocSize, 0, 0, 0));
        CU_ASSERT_RESULT(hipMemMap((hipDeviceptr_t)x, allocSize, 0, handleX, 0));
        CU_ASSERT_RESULT(hipMemMap((hipDeviceptr_t)y, allocSize, 0, handleY, 0));
        // explicitly protect mapped VA ranges
        hipMemAccessDesc accessDesc[2] = {{}};
        accessDesc[0].location.type = hipMemLocationTypeDevice;
        accessDesc[0].location.id = currentDev;
        accessDesc[0].flags = hipMemAccessFlagsProtReadWrite;
        accessDesc[1].location.type = type;
        accessDesc[1].location.id = hostMem ? cpuNumaNodeId : currentDev;
        accessDesc[1].flags = hipMemAccessFlagsProtReadWrite;
        CU_ASSERT_RESULT(hipMemSetAccess((hipDeviceptr_t)x, allocSize, accessDesc, hostMem ? 2 : 1));
        CU_ASSERT_RESULT(hipMemSetAccess((hipDeviceptr_t)y, allocSize, accessDesc, hostMem ? 2 : 1));
    } else if (allocator == "malloc") {
        x = (float *)(malloc(allocSize));
        y = (float *)(malloc(allocSize));
    } else {
        std::cout << "Unknown allocator " << allocator << std::endl;
        return -1;
    }

    float *x_cpu, *y_cpu;
    if (allocator == "hipMemCreate-Device") {
        x_cpu = (float *)(malloc(allocSize));
        y_cpu = (float *)(malloc(allocSize));
        for (size_t i = 0; i < N; ++i) {
            x_cpu[i] = 1.0f;
            y_cpu[i] = 2.0f;
        }
        CU_ASSERT_ERROR(hipMemcpy(x, x_cpu, allocSize, hipMemcpyHostToDevice));
        CU_ASSERT_ERROR(hipMemcpy(y, y_cpu, allocSize, hipMemcpyHostToDevice));
    } else {
        for (size_t i = 0; i < N; ++i) {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }
    }

    hipEvent_t ckpt1, ckpt2, ckpt3;
    CU_ASSERT_ERROR(hipEventCreate(&ckpt1));
    CU_ASSERT_ERROR(hipEventCreate(&ckpt2));
    CU_ASSERT_ERROR(hipEventCreate(&ckpt3));

    std::cout << "Initialized " << N << " elements using " << allocator << std::endl;

    size_t blockSize = 1024;
    size_t numBlocks = (N + blockSize - 1) / blockSize;

    CU_ASSERT_ERROR(hipEventRecord(ckpt1));

    add<<<numBlocks, blockSize>>>(N, x, y);
    CU_ASSERT_ERROR(hipGetLastError());

    CU_ASSERT_ERROR(hipEventRecord(ckpt2));

    for (int k = 0; k < 10; ++k) {
        add<<<numBlocks, blockSize>>>(N, x, y);
        CU_ASSERT_ERROR(hipGetLastError());
    }

    CU_ASSERT_ERROR(hipEventRecord(ckpt3));

    CU_ASSERT_ERROR(hipEventSynchronize(ckpt3));
    // CU_ASSERT_ERROR(hipDeviceSynchronize());

    float elapsed = 0;
    CU_ASSERT_ERROR(hipEventElapsedTime(&elapsed, ckpt1, ckpt2));
    std::cout << "Elapsed time " << int(elapsed) << " mili-seconds - initial" << std::endl;
    CU_ASSERT_ERROR(hipEventElapsedTime(&elapsed, ckpt2, ckpt3));
    std::cout << "Elapsed time " << int(elapsed / 10) << " mili-seconds - warmedup" << std::endl;

    float maxError = 0.0f;
    if (allocator == "hipMemCreate-Device") {
        CU_ASSERT_ERROR(hipMemcpy(y_cpu, y, allocSize, hipMemcpyDeviceToHost));
        for (size_t i = 0; i < N; i++) {
            maxError = fmax(maxError, fabs(y_cpu[i] - 13.0f));
        }
    } else {
        for (size_t i = 0; i < N; i++) {
            maxError = fmax(maxError, fabs(y[i] - 13.0f));
        }
    }

    CU_ASSERT_ERROR(hipEventDestroy(ckpt1));
    CU_ASSERT_ERROR(hipEventDestroy(ckpt2));
    CU_ASSERT_ERROR(hipEventDestroy(ckpt3));

    if (allocator == "hipMemCreate-Device" || allocator == "hipMemCreate-Host") {
        CU_ASSERT_RESULT(hipMemUnmap((hipDeviceptr_t)x, allocSize));
        CU_ASSERT_RESULT(hipMemUnmap((hipDeviceptr_t)y, allocSize));
        CU_ASSERT_RESULT(hipMemAddressFree((hipDeviceptr_t)x, allocSize));
        CU_ASSERT_RESULT(hipMemAddressFree((hipDeviceptr_t)y, allocSize));
        CU_ASSERT_RESULT(hipMemRelease(handleX));
        CU_ASSERT_RESULT(hipMemRelease(handleY));
        if (allocator == "hipMemCreate-Device") {
            free(x_cpu);
            free(y_cpu);
        }
    } else if (allocator == "malloc") {
        free(x);
        free(y);
    } else if (allocator == "hipHostMalloc") {
        CU_ASSERT_ERROR(hipHostFree(x));
        CU_ASSERT_ERROR(hipHostFree(y));
    } else {
        CU_ASSERT_ERROR(hipFree(x));
        CU_ASSERT_ERROR(hipFree(y));
    }
    return int(maxError);
}

int main(int argc, char **argv) {
    std::set<std::string> avail_allocators(
        {"hipHostMalloc", "hipMallocManaged", "hipMemCreate-Device", "hipMemCreate-Host", "malloc"});
    std::string allocator;
    po::options_description all_opts("uvm_vec_add CLI");
    all_opts.add_options()("help,h", "Help message");
    all_opts.add_options()("list,l", "list available allocator options");
    all_opts.add_options()("allocator,a", po::value<std::string>(&allocator)->default_value("all"),
                           "Which memory allocation API to use");

    po::variables_map vm;
    try {
        po::store(po::parse_command_line(argc, argv, all_opts), vm);
        po::notify(vm);
    } catch (...) {
        std::cout << "Error: Invalid Arguments " << std::endl;
        std::cout << all_opts << std::endl;
        return 1;
    }
    if (vm.count("help")) {
        std::cout << all_opts << std::endl;
        return 0;
    }
    if (vm.count("list")) {
        for (const std::string &x : avail_allocators) {
            std::cout << x << std::endl;
        }
        return 0;
    }

    int maxError = 0;
    if (allocator == "all") {
        for (const std::string &x : avail_allocators) {
            std::cout << "\nBenchmarking " << x << std::endl;
            maxError = benchmark(x);
            std::cout << "Max error is " << maxError << std::endl;
        }
    } else {
        maxError = benchmark(allocator);
        std::cout << "Max error is " << maxError << std::endl;
    }

    return 0;
}
