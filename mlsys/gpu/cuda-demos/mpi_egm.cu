#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <nvml.h>
#include <stdexcept>
#include <unistd.h>

// CUDA error handling
#define CU_ASSERT_RESULT(x)                                                                                            \
    do {                                                                                                               \
        hipError_t cuResult = (x);                                                                                       \
        if ((cuResult) != hipSuccess) {                                                                              \
            const char *errDescStr, *errNameStr;                                                                       \
            hipDrvGetErrorString(cuResult, &errDescStr);                                                                   \
            hipDrvGetErrorName(cuResult, &errNameStr);                                                                     \
            fprintf(stderr, "[%s] %s in expr %s in %s() : %s:%d\n", errNameStr, errDescStr, #x, __PRETTY_FUNCTION__,   \
                    __FILE__, __LINE__);                                                                               \
            std::exit(EXIT_FAILURE);                                                                                   \
        }                                                                                                              \
    } while (0)

#define CU_ASSERT_ERROR(x)                                                                                             \
    do {                                                                                                               \
        hipError_t cuError = (x);                                                                                     \
        if ((cuError) != hipSuccess) {                                                                                \
            fprintf(stderr, "[%s] %s in expr %s in %s() : %s:%D\n", hipGetErrorName(cuError),                         \
                    hipGetErrorString(cuError), #x, __PRETTY_FUNCTION__, __FILE__, __LINE__);                         \
            std::exit(EXIT_FAILURE);                                                                                   \
        }                                                                                                              \
    } while (0)

#define NVML_ASSERT(x)                                                                                                 \
    do {                                                                                                               \
        nvmlReturn_t nvmlResult = (x);                                                                                 \
        if ((nvmlResult) != NVML_SUCCESS) {                                                                            \
            fprintf(stderr, "NVML error %d:%s\n", nvmlResult, nvmlErrorString(nvmlResult));                            \
            std::exit(EXIT_FAILURE);                                                                                   \
        }                                                                                                              \
    } while (0)

// GPU kernels
__global__ void init(float *buffer, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < size; i += stride) {
        buffer[i] = static_cast<float>(i);
    }
}

__global__ void reduce(float *input, float *output, int size) {
    extern __shared__ float sdata[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Load data into shared memory, handling out-of-bounds elements
    sdata[tid] = (idx < size) ? input[idx] : 0;
    __syncthreads();

    // Perform tree-based reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0)
        output[blockIdx.x] = sdata[0];
}

int main(int argc, char **argv) {
    char hostname[256];
    gethostname(hostname, sizeof(hostname));

    // Initialize the MPI environment
    MPI_Init(&argc, &argv);

    int world_size, world_rank;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);

    printf("Rank %d on %s out of %d total ranks\n", world_rank, hostname, world_size);

    // cuda initialzation
    CU_ASSERT_RESULT(hipInit(0));
    NVML_ASSERT(nvmlInit());
    MPI_Barrier(MPI_COMM_WORLD);

    // get device handle
    int cudaDev;
    hipDevice_t currentDev;
    CU_ASSERT_ERROR(hipGetDevice(&cudaDev));
    CU_ASSERT_RESULT(hipDeviceGet(&currentDev, cudaDev));
    printf("get cuda device %d/%d\n", cudaDev, currentDev);

    // get cpu NUMA id and set location type
    int cpuNumaNodeId = -1;
    hipMemLocationType location_type = hipMemLocationTypeDevice;
    CU_ASSERT_RESULT(hipDeviceGetAttribute(&cpuNumaNodeId, CU_DEVICE_ATTRIBUTE_HOST_NUMA_ID, currentDev));
    bool hostMem = (cpuNumaNodeId != -1);
    // hostMem = false;
    location_type = hostMem ? hipMemLocationTypeHostNuma : location_type;
    printf("hostMem-%d, host numa ID=%d\n", hostMem, cpuNumaNodeId);

    int vmm = 0;
    CU_ASSERT_RESULT(hipDeviceGetAttribute(&vmm, hipDeviceAttributeVirtualMemoryManagementSupported, currentDev));
    if (!vmm) {
        throw std::runtime_error("Virtual memory management not supported on this device");
    }

    // memory allocation property
    hipMemAllocationProp memprop = {};
    memprop.type = hipMemAllocationTypePinned;
    memprop.requestedHandleTypes = CU_MEM_HANDLE_TYPE_FABRIC;
    memprop.location.type = location_type;
    memprop.location.id = hostMem ? cpuNumaNodeId : currentDev;

    // size & granularity
    size_t allocSize = ((size_t)(256) << 20) * sizeof(float);
    size_t granu = 0;
    CU_ASSERT_RESULT(hipMemGetAllocationGranularity(&granu, &memprop, hipMemAllocationGranularityRecommended));
    size_t origSize = allocSize;
    if (allocSize % granu > 0) {
        allocSize = granu * (allocSize / granu + 1);
    }
    printf("requested size=%zu, padded alloc size=%zu, granularity=%zu\n", origSize, allocSize, granu);

    // allocate physical memory
    hipMemGenericAllocationHandle_t allocHandle;
    CUmemFabricHandle fabricHandle;
    if (world_rank == 0) {
        CU_ASSERT_RESULT(hipMemCreate(&allocHandle, allocSize, &memprop, 0));
        CU_ASSERT_RESULT(hipMemExportToShareableHandle(&fabricHandle, allocHandle, CU_MEM_HANDLE_TYPE_FABRIC, 0));
        printf("allocHandle=%llx\n", allocHandle);
        printf("fabricHandle=");
        for (auto x : fabricHandle.data) {
            printf("%02x", x);
        }
        printf("\n");
    }

    // broadcast fabric handle
    MPI_Bcast(&fabricHandle, sizeof(fabricHandle), MPI_BYTE, 0, MPI_COMM_WORLD);

    if (world_rank != 0) {
        printf("fabricHandle=");
        for (auto x : fabricHandle.data) {
            printf("%02x", x);
        }
        printf("\n");
        CU_ASSERT_RESULT(
            hipMemImportFromShareableHandle(&allocHandle, (void *)&fabricHandle, CU_MEM_HANDLE_TYPE_FABRIC));
        printf("allocHandle=%llx\n", allocHandle);
    }
    // just for safety: wait after every rank is done importing
    MPI_Barrier(MPI_COMM_WORLD);

    // map the memory
    float *buffer_egm;
    CU_ASSERT_RESULT(hipMemAddressReserve((hipDeviceptr_t *)&buffer_egm, allocSize, 0, 0, 0));
    CU_ASSERT_RESULT(hipMemMap((hipDeviceptr_t)buffer_egm, allocSize, 0, allocHandle, 0));

    // explicitly protect mapped VA ranges
    hipMemAccessDesc accessDesc;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    // always make it accessible to the device
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = currentDev;
    CU_ASSERT_RESULT(hipMemSetAccess((hipDeviceptr_t)buffer_egm, allocSize, &accessDesc, 1));
    if (world_rank == 0 && location_type != hipMemLocationTypeDevice) {
        accessDesc.location.type = location_type;
        accessDesc.location.id = hostMem ? cpuNumaNodeId : currentDev;
        CU_ASSERT_RESULT(hipMemSetAccess((hipDeviceptr_t)buffer_egm, allocSize, &accessDesc, 1));
    }

    // Make sure every rank is done with mapping the fabric allocation
    MPI_Barrier(MPI_COMM_WORLD);

    // write data from one rank and read from other ranks
    int writer_rank = 0;
    int reader_rank = 1;

    size_t num_elements = allocSize / sizeof(float);
    int block_size = 256;
    int num_blocks = (num_elements + (size_t)(block_size - 1)) / block_size;
    // printf("num_elements=%zu, block_size=%d, num_blocks=%d\n", num_elements,
    //        block_size, num_blocks);
    // write from writer_rank
    if (world_rank == writer_rank || world_size == 1) {
        printf("Write data from rank=%d to EGM buffer\n", world_rank);
        init<<<num_blocks, block_size>>>(buffer_egm, num_elements);
        CU_ASSERT_ERROR(hipGetLastError());
        CU_ASSERT_ERROR(hipDeviceSynchronize());
    }

    // sync
    printf("Rank %d waiting at barrier\n", world_rank);
    fflush(stdout);
    MPI_Barrier(MPI_COMM_WORLD);
    // extra safety
    CU_ASSERT_ERROR(hipDeviceSynchronize());
    MPI_Barrier(MPI_COMM_WORLD);

    // read from other ranks
    if (world_rank == reader_rank || world_size == 1) {
        float *sum;
        sum = (float *)malloc(sizeof(float) * num_blocks);

        // verify
        printf("Reading from rank=%d to sum up first 10 elements\n", world_rank);
        reduce<<<num_blocks, block_size, block_size * sizeof(float)>>>(buffer_egm, sum, 10);
        CU_ASSERT_ERROR(hipGetLastError());
        CU_ASSERT_ERROR(hipDeviceSynchronize());
        printf("sum[0]=%f\n", sum[0]);

        hipEvent_t ckpt1, ckpt2, ckpt3;
        float elapsed2 = 0, elapsed3 = 0;
        hipEventCreate(&ckpt1);
        hipEventCreate(&ckpt2);
        hipEventCreate(&ckpt3);
        // allocate local gpu buffer
        float *buffer_local;
        CU_ASSERT_ERROR(hipMalloc(&buffer_local, allocSize));
        // copy
        printf("hipMemcpy buffer of size=%zuMB from rank=%d EGM to rank=%d HBM\n", allocSize / 1024 / 1024, 0,
               world_rank);
        hipEventRecord(ckpt1);
        CU_ASSERT_ERROR(hipMemcpy(buffer_local, buffer_egm, allocSize, hipMemcpyDeviceToDevice));
        hipEventRecord(ckpt2);
        CU_ASSERT_ERROR(hipDeviceSynchronize());
        hipEventRecord(ckpt3);
        hipEventElapsedTime(&elapsed2, ckpt1, ckpt2);
        hipEventElapsedTime(&elapsed3, ckpt1, ckpt3);
        printf("Elapsed time %f/%f mili-seconds\n", elapsed2, elapsed3);
        printf("%s-node EGM->HBM copy bandwidth: %f GB/s\n", world_size == 1 ? "Same" : "Cross",
               (allocSize / 1024.0 / 1024 / 1024) / (elapsed3 / 1000.0));
        // verify after copy
        printf("Verifying data after hipMemcpy from rank=%d\n", world_rank);
        reduce<<<num_blocks, block_size, block_size * sizeof(float)>>>(buffer_local, sum, 10);
        CU_ASSERT_ERROR(hipGetLastError());
        CU_ASSERT_ERROR(hipDeviceSynchronize());
        printf("sum[0]=%f\n", sum[0]);
        // reference copy
        float *buffer_cpu;
        buffer_cpu = (float *)malloc(allocSize);
        for (size_t i = 0; i < num_elements; i++) {
            buffer_cpu[i] = static_cast<float>(i);
        }
        printf("hipMemcpy buffer of size=%zuMB from Host to Device\n", allocSize / 1024 / 1024);
        hipEventRecord(ckpt1);
        CU_ASSERT_ERROR(hipMemcpy(buffer_local, buffer_cpu, allocSize, hipMemcpyHostToDevice));
        hipEventRecord(ckpt2);
        CU_ASSERT_ERROR(hipDeviceSynchronize());
        hipEventRecord(ckpt3);
        hipEventElapsedTime(&elapsed2, ckpt1, ckpt2);
        hipEventElapsedTime(&elapsed3, ckpt1, ckpt3);
        printf("Elapsed time %f/%f mili-seconds\n", elapsed2, elapsed3);
        printf("Same-node H2D copy bandwidth: %f GB/s\n", (allocSize / 1024.0 / 1024 / 1024) / (elapsed3 / 1000.0));
        CU_ASSERT_ERROR(hipFree(buffer_local));
        free(buffer_cpu);
    }

    // cleanup
    MPI_Barrier(MPI_COMM_WORLD);

    CU_ASSERT_RESULT(hipMemUnmap((hipDeviceptr_t)buffer_egm, allocSize));
    CU_ASSERT_RESULT(hipMemRelease(allocHandle));
    CU_ASSERT_RESULT(hipMemAddressFree((hipDeviceptr_t)buffer_egm, allocSize));

    MPI_Finalize();

    return 0;
}
