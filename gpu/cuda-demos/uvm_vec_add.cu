#include <cmath>
#include <cstdint>
#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function to add the elements of two arrays
__global__ void add(size_t n, float *x, float *y)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}

int main(void)
{
    size_t N = (size_t)(2) << 30; // 2G x 4B x 2
    float *x, *y;

    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    for (size_t i = 0; i < N; ++i)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipEvent_t ckpt1, ckpt2, ckpt3;
    hipEventCreate(&ckpt1);
    hipEventCreate(&ckpt2);
    hipEventCreate(&ckpt3);

    std::cout << "Initialized " << N << " elements, GPU kernel start" << std::endl;

    size_t blockSize = 1024;
    size_t numBlocks = (N + blockSize - 1) / blockSize;

    hipEventRecord(ckpt1);

    add<<<numBlocks, blockSize>>>(N, x, y);
    hipEventRecord(ckpt2);

    for (int k = 0; k < 100; ++k) {
        add<<<numBlocks, blockSize>>>(N, x, y);
    }
    hipEventRecord(ckpt3);

    hipDeviceSynchronize();
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, ckpt1, ckpt2);
    std::cout << "Elapsed time " << elapsed / 1.0e3 << " seconds - initial" << std::endl;
    hipEventElapsedTime(&elapsed, ckpt2, ckpt3);
    std::cout << "Elapsed time " << elapsed / 1.0e5 << " seconds - warmedup" << std::endl;

    float maxError = 0.0f;
    for (size_t i = 0; i < N; i++)
    {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }

    hipEventDestroy(ckpt1);
    hipEventDestroy(ckpt2);
    hipEventDestroy(ckpt3);

    hipFree(x);
    hipFree(y);

    return int(maxError);
}